#include <stdio.h>
#include <omp.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>


// device code
__global__
void diffusion(float** u, float r, int size, int iter) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	for (int time = 0; time < iter; time++) {
		float upper, lower, left, right;
		if (i != 0 && i != (size - 1) && j != 0 && j != (size - 1)) {
			upper = u[i - 1][j];
			lower = u[i + 1][j];
			left  = u[i][j - 1];
			right = u[i][j + 1];
			__syncthreads();
			u[i][j] = (1 - 4 * r) * u[i][j] + r * (upper + lower + left + right);
		}
		__syncthreads();
	}
}

//host code
__host__
int main() {
    struct timeval tv_before, tv_after;
	int size = 100;
	int iter = 100;
	float u[100][100] = {0};
	float out[100][100] = {0};
	float r = 0.2;
	for (int i = 1; i < size - 1; i++) {
		for (int j = 1; j < size - 1; j++) {
			u[i][j] = 1.0;
			out[i][j] = 0.5;
		}
	}
	float **address;
	size_t pitch;
	size_t host_pitch = size * sizeof(float);

	hipMallocPitch((void**)&address, &pitch, size * sizeof(float), size);
	printf("%d, %d\n", (int)pitch, (int)host_pitch);
	//cudaMemcpy2D(address, host_pitch, u, pitch, size * sizeof(float), size, cudaMemcpyHostToDevice);
	hipMemcpy2D(address, pitch, u, host_pitch, size * sizeof(float), size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(size, size);
	dim3 numBlocks(2, 2);
	
	gettimeofday(&tv_before, NULL);
	diffusion<<<numBlocks, threadsPerBlock>>>(address, r, size, iter);
	gettimeofday(&tv_after, NULL);
	
	//cudaMemcpy2D(u, pitch, address, host_pitch, size * sizeof(float), size, cudaMemcpyDeviceToHost);
	hipMemcpy2D(out, host_pitch, address, pitch, size * sizeof(float), size, hipMemcpyDeviceToHost);
	hipFree(address);
	
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			int color = (int)(out[i][j] * 255);
			printf("\033[48;2;%d;%d;255m  ", color, color);
		}
		printf("\033[0m\n");
	}
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%.3f ", out[i][j]);
		}
		printf("\033[0m\n");
	}

	printf("\033[0m\n");
	printf("time : %ld sec + %06lu usec\n", 
			tv_after.tv_sec - tv_before.tv_sec, 
			tv_after.tv_usec - tv_before.tv_usec);
}
