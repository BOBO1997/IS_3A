#include <stdio.h>
#include <omp.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>


// device code
__global__
void diffusion(float* u, size_t pitch, float r, int size, int iter) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	for (int time = 0; time < iter; time++) {
		if (i != 0 && i != (size - 1) && j != 0 && j != (size - 1)) {
			float upper, lower, left, right;
			float* row = (float*)((char*)u + j * pitch);
			float* upper_row = (float*)((char*)u + (j - 1) * pitch);
			float* lower_row = (float*)((char*)u + (j + 1) * pitch);
			upper = upper_row[i];
			lower = lower_row[i];
			left  = row[i - 1];
			right = row[i + 1];
			__syncthreads();
			row[i] = (1 - 4 * r) * row[i] + r * (upper + lower + left + right);
		}
		__syncthreads();
	}
}

//host code
__host__
int main() {
    struct timeval tv_before, tv_after;
	int size = 512;
	int iter = 100;
	int thread_width = 32;
	float u[512][512] = {0};
	float out[512][512] = {0};
	float r = 0.225;
	for (int i = 1; i < size - 1; i++) {
		for (int j = 1; j < size - 1; j++) {
			u[i][j] = 1.0;
		}
	}
	//float **address;
	float *address;
	size_t pitch;
	size_t host_pitch = size * sizeof(float);

	hipMallocPitch(&address, &pitch, size * sizeof(float), size);
	printf("host_pitch = %d, device_pitch = %d\n", (int)host_pitch, (int)pitch);
	hipMemcpy2D(address, pitch, u, host_pitch, size * sizeof(float), size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(thread_width, thread_width);
	dim3 numBlocks(size / thread_width, size / thread_width);
	
	gettimeofday(&tv_before, NULL);
	diffusion<<<numBlocks, threadsPerBlock>>>(address, pitch, r, size, iter);
	hipDeviceSynchronize();
	gettimeofday(&tv_after, NULL);
	
	hipMemcpy2D(out, host_pitch, address, pitch, size * sizeof(float), size, hipMemcpyDeviceToHost);
	hipFree(address);
	
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			int color = (int)(out[i][j] * 255);
			printf("\033[48;2;%d;%d;255m ", color, color);
		}
		printf("\033[0m\n");
	}
	/*
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%.3f ", out[i][j]);
		}
		printf("\033[0m\n");
	}
	*/
	printf("\033[0m\n");
	printf("time : %06ld sec + %lu usec\n", 
			tv_after.tv_sec - tv_before.tv_sec, 
			tv_after.tv_usec - tv_before.tv_usec);
}
