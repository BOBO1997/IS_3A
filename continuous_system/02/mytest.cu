#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Kernel definition
/*
__global__ void MatAdd(int N, float A[N][N], float B[N][N],
float C[N][N])
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N)
		C[i][j] = A[i][j] + B[i][j];
}
*/

__host__
int main()
{
	int N = 16;
	int A[16][16], B[16][16], C[16][16];
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			A[i][j] = i + j;
			B[i][j] = i - j;
			C[i][j] = i * j;
		}
	}
	hipMallocPitch(());
	
	// Kernel invocation
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
	MatAdd<<<numBlocks, threadsPerBlock>>>(N, A, B, C);
}
