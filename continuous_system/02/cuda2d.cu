#include<stdio.h>
#include<hip/hip_runtime.h>
#define height 50
#define width 50

// Device code
__global__ void kernel(float* devPtr, int pitch)
{
    for (int r = 0; r < height; ++r) {
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) {
             float element = row[c];
        }
    }
}

//Host Code
int main()
{
	float* devPtr;
	size_t pitch;
	hipMallocPitch((void**)&devPtr, &pitch, width * sizeof(float), height);
	printf("%d\n", (int)pitch);
	kernel<<<100, 512>>>(devPtr, pitch);
	return 0;
}
