#include <stdio.h>
#include <omp.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>


// device code
__global__
void diffusion(float* u, float r, int size, int iter) {
	
	int i = threadIdx.x / size;
	int j = threadIdx.x % size;
	for (int time = 0; time < iter; time++) {
		if (i != 0 && i != (size - 1) && j != 0 && j != (size - 1)) {
			float upper = u[threadIdx.x - size];
			float lower = u[threadIdx.x + size];
			float left  = u[threadIdx.x - 1];
			float right = u[threadIdx.x + 1];
			__syncthreads();
			u[threadIdx.x] = (1 - 4 * r) * u[threadIdx.x] + r * (upper + lower + left + right);
		}
		__syncthreads();
	}
}

//host code
__host__
int main() {
    struct timeval tv_before, tv_after;
	int size = 30;
	int iter = 100;
	float u[30 * 30] = {0};
	float out[30 * 30] = {0};
	float r = 0.2;
	for (int i = 1; i < size - 1; i++) {
		for (int j = 1; j < size - 1; j++) {
			u[i * size + j] = 1.0;
		}
	}
	float *address;
	const int fsize = size * size * sizeof(float);
	hipMalloc((void**)&address, fsize);
	hipMemcpy(address, u, fsize, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(size * size, 1);
	dim3 numBlocks(1, 1);
	
	gettimeofday(&tv_before, NULL);
	diffusion<<<numBlocks, threadsPerBlock>>>(address, r, size, iter);
	hipDeviceSynchronize();
	gettimeofday(&tv_after, NULL);

	hipMemcpy(out, address, fsize, hipMemcpyDeviceToHost);
	hipFree(address);
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			int color = (int)(out[i * size + j] * 255);
			printf("\033[48;2;%d;%d;255m  ", color, color);
		}
		printf("\033[0m\n");
	}
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%.3f ", out[i * size + j]); //$B?tCM$r(Bdump
		}
		printf("\033[0m\n");
	}
	
	printf("\033[0m\n");
	printf("time : %ld sec + %06lu usec\n", 
			tv_after.tv_sec - tv_before.tv_sec, 
			tv_after.tv_usec - tv_before.tv_usec);
}
