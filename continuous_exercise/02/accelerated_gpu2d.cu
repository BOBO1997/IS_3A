#include <stdio.h>
#include <omp.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>


// device code
__global__
void diffusion(float* even, float* odd, size_t pitch, float r, int size, int iter) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	for (int time = 0; time < iter; time++) {
		if (i != 0 && i != (size - 1) && j != 0 && j != (size - 1)) {
			if (time % 2 == 0) {
				float* even_row = (float*)((char*)even + j * pitch);
				float* odd_row = (float*)((char*)odd + j * pitch);
				float* upper_row = (float*)((char*)even + (j - 1) * pitch);
				float* lower_row = (float*)((char*)even + (j + 1) * pitch);
				float upper = upper_row[i];
				float lower = lower_row[i];
				float left  = even_row[i - 1];
				float right = even_row[i + 1];
				odd_row[i] = (1 - 4 * r) * even_row[i] + r * (upper + lower + left + right);
			}
			else {
				float* odd_row = (float*)((char*)odd + j * pitch);
				float* even_row = (float*)((char*)even + j * pitch);
				float* upper_row = (float*)((char*)odd + (j - 1) * pitch);
				float* lower_row = (float*)((char*)odd + (j + 1) * pitch);
				float upper = upper_row[i];
				float lower = lower_row[i];
				float left  = odd_row[i - 1];
				float right = odd_row[i + 1];
				even_row[i] = (1 - 4 * r) * odd_row[i] + r * (upper + lower + left + right);
			}
		}
		__syncthreads();
	}
}

//host code
__host__
int main() {
    struct timeval tv_before, tv_after;
	int size = 512;
	int iter = 100;
	int thread_width = 32;
	float even[512][512] = {0};
	float odd[512][512] = {0};
	float out[512][512] = {0};
	float r = 0.225;
	for (int i = 1; i < size - 1; i++) {
		for (int j = 1; j < size - 1; j++) {
			even[i][j] = 1.0;
		}
	}
	float *even_ad, *odd_ad;
	size_t pitch;
	size_t host_pitch = size * sizeof(float);

	hipMallocPitch(&even_ad, &pitch, size * sizeof(float), size);
	hipMallocPitch(&odd_ad, &pitch, size * sizeof(float), size);
	printf("host_pitch = %d, device_pitch = %d\n", (int)host_pitch, (int)pitch);
	hipMemcpy2D(even_ad, pitch, even, host_pitch, size * sizeof(float), size, hipMemcpyHostToDevice);
	hipMemcpy2D(odd_ad, pitch, odd, host_pitch, size * sizeof(float), size, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(thread_width, thread_width);
	dim3 numBlocks(size / thread_width, size / thread_width);
	
	gettimeofday(&tv_before, NULL);
	diffusion<<<numBlocks, threadsPerBlock>>>(even_ad, odd_ad, pitch, r, size, iter);
	hipDeviceSynchronize();
	gettimeofday(&tv_after, NULL);
	
	hipMemcpy2D(out, host_pitch, even_ad, pitch, size * sizeof(float), size, hipMemcpyDeviceToHost);
	hipFree(even_ad);
	hipFree(odd_ad);
	/*
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			int color = (int)(out[i][j] * 255);
			printf("\033[48;2;%d;%d;255m ", color, color);
		}
		printf("\033[0m\n");
	}
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%.3f ", out[i][j]);
		}
		printf("\033[0m\n");
	}
	printf("\033[0m\n");
	*/
	printf("time : %ld sec + %06lu usec\n", 
			tv_after.tv_sec - tv_before.tv_sec, 
			tv_after.tv_usec - tv_before.tv_usec);
}
